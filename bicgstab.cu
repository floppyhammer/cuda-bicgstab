#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <stdio.h>
#include <vector>

/*
// Visual Studio
In "Project/Properties/Linker/Input" add "cublas.lib; cusparse.lib;"

// Linux
nvcc -Xcompiler '-fPIC' -lcublas -lcusparse -shared -o libbicgstab.so bicgstab.cu
*/

hipsparseHandle_t cusparseHandle;
hipblasHandle_t cublasHandle;

/// Used for extra memory use in LU decomposition.
void *pBuffer;

/// Set up descriptor for A.
void setUpDescriptor(hipsparseMatDescr_t &descrA, hipsparseMatrixType_t matrixType, hipsparseIndexBase_t indexBase) {
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, matrixType);
    hipsparseSetMatIndexBase(descrA, indexBase);
}

/// Set up descriptor for LU.
void setUpDescriptorLU(hipsparseMatDescr_t &descrLU, hipsparseMatrixType_t matrixType,
                       hipsparseIndexBase_t indexBase, hipsparseFillMode_t fillMode,
                       hipsparseDiagType_t diagType) {
    hipsparseCreateMatDescr(&descrLU);
    hipsparseSetMatType(descrLU, matrixType);
    hipsparseSetMatIndexBase(descrLU, indexBase);
    hipsparseSetMatFillMode(descrLU, fillMode);
    hipsparseSetMatDiagType(descrLU, diagType);
}

/// Memory query for LU.
void memoryQueryLU(csrilu02Info_t &infoA, csrsv2Info_t &infoL, csrsv2Info_t &infoU,
                   hipsparseHandle_t cusparseHandle, const int n, const int nnz,
                   hipsparseMatDescr_t &descrA, hipsparseMatDescr_t &descrL, hipsparseMatDescr_t &descrU,
                   double *d_A, const int *d_A_RowPtr, const int *d_A_ColInd,
                   hipsparseOperation_t matrixOperation, void **pBuffer) {
    hipsparseCreateCsrilu02Info(&infoA);
    hipsparseCreateCsrsv2Info(&infoL);
    hipsparseCreateCsrsv2Info(&infoU);

    int pBufferSize_M, pBufferSize_L, pBufferSize_U;
    hipsparseDcsrilu02_bufferSize(cusparseHandle, n, nnz, descrA, d_A, d_A_RowPtr,
                                 d_A_ColInd, infoA, &pBufferSize_M);
    hipsparseDcsrsv2_bufferSize(cusparseHandle, matrixOperation, n, nnz, descrL,
                               d_A, d_A_RowPtr, d_A_ColInd, infoL, &pBufferSize_L);
    hipsparseDcsrsv2_bufferSize(cusparseHandle, matrixOperation, n, nnz, descrU,
                               d_A, d_A_RowPtr, d_A_ColInd, infoU, &pBufferSize_U);

    int pBufferSize = std::max(pBufferSize_M, std::max(pBufferSize_L, pBufferSize_U));

    hipMalloc((void **) pBuffer, pBufferSize);
}

/// Analysis for LU.
void analyzeLU(csrilu02Info_t &infoA, csrsv2Info_t &infoL,
               csrsv2Info_t &infoU, hipsparseHandle_t cusparseHandle, const int N,
               const int nnz, hipsparseMatDescr_t descrA, hipsparseMatDescr_t &descrL,
               hipsparseMatDescr_t &descrU, double *d_A, const int *d_A_RowPtr,
               const int *d_A_ColInd, hipsparseOperation_t matrixOperation,
               hipsparseSolvePolicy_t solvePolicy1, hipsparseSolvePolicy_t solvePolicy2,
               void *pBuffer) {
    int structural_zero;

    hipsparseDcsrilu02_analysis(cusparseHandle, N, nnz, descrA, d_A, d_A_RowPtr,
                               d_A_ColInd, infoA, solvePolicy1, pBuffer);

    hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(cusparseHandle, infoA, &structural_zero);

    if (HIPSPARSE_STATUS_ZERO_PIVOT == status) {
        printf("A(%d, %d) is missing\n", structural_zero, structural_zero);
    }

    hipsparseDcsrsv2_analysis(cusparseHandle, matrixOperation, N, nnz, descrL,
                             d_A, d_A_RowPtr, d_A_ColInd, infoL, solvePolicy1, pBuffer);
    hipsparseDcsrsv2_analysis(cusparseHandle, matrixOperation, N, nnz, descrU,
                             d_A, d_A_RowPtr, d_A_ColInd, infoU, solvePolicy2, pBuffer);
}

/// Incomplete LU decomposition.
void computeLU(csrilu02Info_t &infoA, hipsparseHandle_t cusparseHandle,
               const int N, const int nnz, hipsparseMatDescr_t &descrA,
               double *d_A, const int *d_A_RowPtr, const int *d_A_ColInd,
               hipsparseSolvePolicy_t solutionPolicy, void *pBuffer) {
    int numericalZero;

    hipsparseDcsrilu02(cusparseHandle, N, nnz, descrA, d_A, d_A_RowPtr, d_A_ColInd,
                      infoA, solutionPolicy, pBuffer);

    hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(cusparseHandle, infoA,
                                                          &numericalZero);

    if (HIPSPARSE_STATUS_ZERO_PIVOT == status) {
        printf("U(%d, %d) is zero\n", numericalZero, numericalZero);
    }
}

void getIncompleteLU(hipsparseHandle_t &cusparseHandle, hipsparseMatDescr_t &descrA,
                     hipsparseMatDescr_t &descrL, hipsparseMatDescr_t &descrU, csrilu02Info_t &infoA,
                     csrsv2Info_t &infoL, csrsv2Info_t &infoU, int n, int nnz, double *valACopy,
                     const int *rowPtr, const int *colInd) {
    // Step 1: Set up descriptors for A, L and U.
    setUpDescriptor(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ZERO);
    setUpDescriptorLU(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ZERO,
                      HIPSPARSE_FILL_MODE_LOWER, HIPSPARSE_DIAG_TYPE_UNIT);
    setUpDescriptorLU(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ZERO,
                      HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    // Step 2: Query how much memory used in LU factorization and the two following system inversions.
    memoryQueryLU(infoA, infoL, infoU, cusparseHandle, n, nnz, descrA, descrL, descrU,
                  valACopy, rowPtr, colInd, HIPSPARSE_OPERATION_NON_TRANSPOSE, &pBuffer);

    // Step 3: Analyze the three problems: LU factorization and the two following system inversions.
    analyzeLU(infoA, infoL, infoU, cusparseHandle, n, nnz, descrA, descrL, descrU,
              valACopy, rowPtr, colInd, HIPSPARSE_OPERATION_NON_TRANSPOSE,
              HIPSPARSE_SOLVE_POLICY_NO_LEVEL, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);

    // Step 4: Factorize A = L * U (A will be overwritten).
    computeLU(infoA, cusparseHandle, n, nnz, descrA, valACopy, GRowPtr, GColInd,
              HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);
}

void spSolverBiCGStab(int n, int nnz, const double *valA, const int *rowPtr, const int *colInd,
                      const double *b, double *x, double tol) {
    // Create descriptors for A, L and U.
    hipsparseMatDescr_t descrA, descrL, descrU;

    // Create ILU and SV info for A, L and U.
    csrilu02Info_t infoA;
    csrsv2Info_t infoL, infoU;

    // Create a copy of A for incomplete LU decomposition.
    // This copy will be modified in the solving process.
    double *valACopy;
    hipMalloc((void **) &valACopy, nnz * sizeof(double));
    hipMemcpy(valACopy, valA, nnz * sizeof(double), hipMemcpyDeviceToDevice);

    // Incomplete LU.
    getIncompleteLU(cusparseHandle, descrA, descrL, descrU, infoA, infoL, infoU, n, nnz, valACopy, rowPtr, colInd);

    double *r;
    hipMalloc((void **) &r, n * sizeof(double));
    double *rw;
    hipMalloc((void **) &rw, n * sizeof(double));
    double *p;
    hipMalloc((void **) &p, n * sizeof(double));
    double *ph;
    hipMalloc((void **) &ph, n * sizeof(double));
    double *t;
    hipMalloc((void **) &t, n * sizeof(double));
    double *q;
    hipMalloc((void **) &q, n * sizeof(double));
    double *s;
    hipMalloc((void **) &s, n * sizeof(double));

    double one = 1, nega_one = -1, zero = 0;
    double alpha, negalpha, beta, omega, nega_omega;
    double temp1, temp2;
    double rho = 0.0, rhop;
    double nrmr0, nrmr;
    int niter = 0;

    // Initial guess x0 (all zeros here).
    hipblasDscal(cublasHandle, n, &zero, x, 1);

    // 1: compute the initial residual r = b - A * x0.
    hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &nega_one, descrA, valA, rowPtr,
                   colInd, x, &zero, r);
    hipblasDaxpy(cublasHandle, n, &one, b, 1, r, 1);

    // 2: copy r into rw and p.
    hipblasDcopy(cublasHandle, n, r, 1, rw, 1);
    hipblasDcopy(cublasHandle, n, r, 1, p, 1);

    //hipblasDnrm2(cublasHandle, n, r, 1, &nrmr0);
    //hipDeviceSynchronize();

    // Repeat until convergence.
    while (true) {
        rhop = rho;
        hipblasDdot(cublasHandle, n, rw, 1, r, 1, &rho);

        if (niter > 0) {
            // 12
            beta = (rho / rhop) * (alpha / omega);

            // 13, p = r + beta * (p - omega * v)
            hipblasDaxpy(cublasHandle, n, &nega_omega, q, 1, p, 1);  // p += -omega * v
            hipblasDscal(cublasHandle, n, &beta, p, 1);  // p *= beta
            hipblasDaxpy(cublasHandle, n, &one, r, 1, p, 1);  // p += 1 * r
        }

        // 15: solve M * pw = p for pw.
        hipsparseDcsrsv2_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz, &one, descrL, valACopy, rowPtr,
                              colInd, infoL, p, t, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);
        hipsparseDcsrsv2_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz, &one, descrU, valACopy, rowPtr,
                              colInd, infoU, t, ph, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);

        // 16
        hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descrA, valA, rowPtr, colInd,
                       ph, &zero, q);

        // 17
        hipblasDdot(cublasHandle, n, rw, 1, q, 1, &temp1);
        hipDeviceSynchronize();
        alpha = rho / temp1;
        negalpha = -alpha;

        // 18
        hipblasDaxpy(cublasHandle, n, &negalpha, q, 1, r, 1);

        // 19
        hipblasDaxpy(cublasHandle, n, &alpha, ph, 1, x, 1);

        // 20
        hipblasDnrm2(cublasHandle, n, r, 1, &nrmr);
        hipDeviceSynchronize();
        if (nrmr < tol) break;

        // 23: solve M * sh = r for sh, note that s is sh for now.
        hipsparseDcsrsv2_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz, &one, descrL, valACopy, rowPtr,
                              colInd, infoL, r, t, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);
        hipsparseDcsrsv2_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz, &one, descrU, valACopy, rowPtr,
                              colInd, infoU, t, s, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);

        // 24
        hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descrA, valA, rowPtr, colInd,
                       s, &zero, t);

        // 25: omega = np.dot(t, r) / np.dot(t, t).
        hipblasDdot(cublasHandle, n, t, 1, r, 1, &temp1);
        hipblasDdot(cublasHandle, n, t, 1, t, 1, &temp2);
        hipDeviceSynchronize();
        omega = temp1 / temp2;
        nega_omega = -omega;

        // 26
        hipblasDaxpy(cublasHandle, n, &omega, s, 1, x, 1);
        hipblasDaxpy(cublasHandle, n, &nega_omega, t, 1, r, 1);

        hipblasDnrm2(cublasHandle, n, r, 1, &nrmr);
        hipDeviceSynchronize();
        if (nrmr < tol) break;

        niter++;

        //printf("Norm: %f\n", nrmr);
    }

    //printf("Number of iterations: %d\n", niter);

    // Clean up
    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroyMatDescr(descrL);
    hipsparseDestroyMatDescr(descrU);
    hipsparseDestroyCsrilu02Info(infoA);
    hipsparseDestroyCsrsv2Info(infoL);
    hipsparseDestroyCsrsv2Info(infoU);
    hipFree(r);
    hipFree(rw);
    hipFree(p);
    hipFree(ph);
    hipFree(t);
    hipFree(q);
    hipFree(s);
    hipFree(valACopy);
    hipFree(pBuffer);;
}

extern "C" {
// Add __declspec(dllexport) if using Visual Studio.
__declspec(dllexport) void solve(int *rowPtr, int *colInd, double *csrData,
                                 double *b, double *x, int n, int nnz, double tol) {
    // Allocate GPU memory
    // ------------------------------------------
    // Copy CSR column indices to GPU.
    int *gColInd;
    hipMalloc((void **) &gColInd, nnz * sizeof(int));
    hipMemcpy(gColInd, colInd, nnz * sizeof(int), hipMemcpyHostToDevice);

    // Copy CSR row offsets to GPU.
    int *gRowPtr;
    hipMalloc((void **) &gRowPtr, (n + 1) * sizeof(int));
    hipMemcpy(gRowPtr, rowPtr, (n + 1) * sizeof(int), hipMemcpyHostToDevice);

    // Copy CSR data array to GPU.
    double *gCsrData;
    hipMalloc((void **) &gCsrData, nnz * sizeof(double));
    hipMemcpy(gCsrData, csrData, nnz * sizeof(double), hipMemcpyHostToDevice);

    // Residual vector.
    double *gB;
    hipMalloc((void **) &gB, n * sizeof(double));
    hipMemcpy(gB, b, n * sizeof(double), hipMemcpyHostToDevice);

    // Solution.
    double *gX;
    hipMalloc((void **) &gX, n * sizeof(double));
    // ------------------------------------------

    // Create CUDA handles.
    hipsparseCreate(&cusparseHandle);
    hipblasCreate(&cublasHandle);

    // Solve Ax = b for x.
    spSolverBiCGStab(n, nnz, gCsrData, gRowPtr, gColInd, gB, gX, tol);

    // Copy x back to CPU.
    hipMemcpy(x, gX, n * sizeof(double), hipMemcpyDeviceToHost);

    // Clean up.
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    hipFree(gX);
    hipFree(gB);

    hipFree(gCsrData);
    hipFree(gColInd);
    hipFree(gRowPtr);
}
}

int main() {
    // Sparse linear matrix in CSR.
    std::vector<int> rowPtr{0, 2, 4, 6, 10, 13};
    std::vector<int> colInd{0, 4, 1, 3, 2, 3, 1, 2, 3, 4, 0, 3, 4};
    std::vector<double> csrData{1., 0.36494769, 1., 0.36768485, 1., 0.34217041,
                                0.36768485, 0.34217041, 1., 0.61652355, 0.36494769, 0.61652355,
                                2.36724823};

    // Right-Hand vector.
    std::vector<double> b{0.66419739, 0.33993935, 0.31049594, 0.78174978, 0.36146131};

    // Solution to calculate.
    std::vector<double> x{0, 0, 0, 0, 0};

    // Vector size.
    int n = 5;

    // None-Zero count.
    int nnz = 13;

    // Tolerance.
    double tol = 1e-8;

    solve(rowPtr.data(), colInd.data(), csrData.data(), b.data(), x.data(), n, nnz, tol);

    return 0;
}
